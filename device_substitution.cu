
#include <factorize.cuh>
#include <comm-mpi.hpp>

void compute_forward_substitution(deviceHandle_t handle, deviceMatrixDesc_t A, const CUDA_CTYPE* X, const ColCommMPI& comm, const ncclComms nccl_comms) {
  long long bdim = A.bdim;
  long long rank = A.rank;
  long long rdim = bdim - rank;
  long long block = bdim * bdim;

  long long D = A.diag_offset;
  long long M = comm.lenLocal();
  long long N = comm.lenNeighbors();
  long long lenA = comm.ARowOffsets[M];
  long long reduc_len = A.reducLen;

  hipStream_t stream = handle->compute_stream;
  hipblasHandle_t cublasH = handle->cublasH;

  STD_CTYPE constants[3] = { 1., 0., -1. };
  CUDA_CTYPE& one = reinterpret_cast<CUDA_CTYPE&>(constants[0]);
  CUDA_CTYPE& zero = reinterpret_cast<CUDA_CTYPE&>(constants[1]); 
  CUDA_CTYPE& minus_one = reinterpret_cast<CUDA_CTYPE&>(constants[2]); 

  hipblasZgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, bdim, 1, bdim, &one, A.Vdata, bdim, block, &X[A.lower_offset], bdim, bdim, &zero, &(A.Ydata)[D * bdim], bdim, bdim, M);

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * bdim;
      long long len = comm.BoxOffsets[p + 1] * bdim - start;
      auto neighbor = nccl_comms->find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Ydata)[start]), &(A.Ydata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms->find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Ydata), A.Ydata, bdim * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }

  size_t sizeX = rank * sizeof(CUDA_CTYPE);
  hipMemcpy2DAsync(&(A.Xdata)[D * rank], sizeX, &(A.Ydata)[D * bdim], bdim * sizeof(CUDA_CTYPE), sizeX, M, hipMemcpyDeviceToDevice, stream);
  if (0 < rank && 0 < rdim) {
    hipMemsetAsync(A.ACdata, 0, reduc_len * M * rank * sizeof(CUDA_CTYPE), stream);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, 1, rdim, &minus_one, A.A_sr, bdim, A.Y_R_cols, bdim, &zero, A.AC_X, rank, lenA);
    hipblasZgemv(cublasH, HIPBLAS_OP_N, M * rank, reduc_len, &one, A.ACdata, M * rank, A.ONEdata, 1, &one, &(A.Xdata)[D * rank], 1);
  }

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * rank;
      long long len = comm.BoxOffsets[p + 1] * rank - start;
      auto neighbor = nccl_comms->find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Xdata)[start]), &(A.Xdata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms->find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Xdata), A.Xdata, rank * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }
}

void compute_backward_substitution(deviceHandle_t handle, deviceMatrixDesc_t A, CUDA_CTYPE* X, const ColCommMPI& comm, const ncclComms nccl_comms) {
  long long bdim = A.bdim;
  long long rank = A.rank;
  long long rdim = bdim - rank;
  long long block = bdim * bdim;

  long long D = A.diag_offset;
  long long M = comm.lenLocal();
  long long N = comm.lenNeighbors();
  long long lenA = comm.ARowOffsets[M];
  long long reduc_len = A.reducLen;

  hipStream_t stream = handle->compute_stream;
  hipblasHandle_t cublasH = handle->cublasH;

  STD_CTYPE constants[3] = { 1., 0., -1. };
  CUDA_CTYPE& one = reinterpret_cast<CUDA_CTYPE&>(constants[0]);
  CUDA_CTYPE& zero = reinterpret_cast<CUDA_CTYPE&>(constants[1]); 
  CUDA_CTYPE& minus_one = reinterpret_cast<CUDA_CTYPE&>(constants[2]); 

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * rank;
      long long len = comm.BoxOffsets[p + 1] * rank - start;
      auto neighbor = nccl_comms->find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Xdata)[start]), &(A.Xdata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms->find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Xdata), A.Xdata, rank * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }

  size_t sizeX = rank * sizeof(CUDA_CTYPE);
  hipMemcpy2DAsync(&(A.Ydata)[D * bdim], bdim * sizeof(CUDA_CTYPE), &(A.Xdata)[D * rank], sizeX, sizeX, M, hipMemcpyDeviceToDevice, stream);
  if (0 < rank && 0 < rdim) {
    hipMemsetAsync(A.ACdata, 0, reduc_len * M * bdim * sizeof(CUDA_CTYPE), stream);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rdim, 1, rank, &minus_one, A.A_rs, bdim, A.X_cols, bdim, &zero, A.AC_X_R, bdim, lenA);
    hipblasZgemv(cublasH, HIPBLAS_OP_N, M * bdim, reduc_len, &one, A.ACdata, M * bdim, A.ONEdata, 1, &one, &(A.Ydata)[D * bdim], 1);
  }

  hipblasZgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_C, 1, bdim, bdim, &one, &(A.Ydata)[D * bdim], 1, bdim, &(A.Udata)[D * block], bdim, block, &zero, &X[A.lower_offset], 1, bdim, M);
}
