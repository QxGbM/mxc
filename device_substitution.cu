
#include <factorize.cuh>
#include <comm-mpi.hpp>

void compute_forward_substitution(deviceMatrixDesc_t A, const CUDA_CTYPE* X, hipStream_t stream, hipblasHandle_t cublasH, const ColCommMPI& comm, const std::map<const MPI_Comm, ncclComm_t>& nccl_comms) {
  long long bdim = A.bdim;
  long long rank = A.rank;
  long long rdim = bdim - rank;
  long long block = bdim * bdim;

  long long D = A.diag_offset;
  long long M = comm.lenLocal();
  long long N = comm.lenNeighbors();
  long long lenA = comm.ARowOffsets[M];
  long long reduc_len = A.reducLen;

  STD_CTYPE constants[3] = { 1., 0., -1. };
  CUDA_CTYPE& one = reinterpret_cast<CUDA_CTYPE&>(constants[0]);
  CUDA_CTYPE& zero = reinterpret_cast<CUDA_CTYPE&>(constants[1]); 
  CUDA_CTYPE& minus_one = reinterpret_cast<CUDA_CTYPE&>(constants[2]); 

  hipblasZgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, bdim, 1, bdim, &one, A.Vdata, bdim, block, &X[A.lower_offset], bdim, bdim, &zero, &(A.Ydata)[D * bdim], bdim, bdim, M);

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * bdim;
      long long len = comm.BoxOffsets[p + 1] * bdim - start;
      auto neighbor = nccl_comms.find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Ydata)[start]), &(A.Ydata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms.find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Ydata), A.Ydata, bdim * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }

  size_t sizeX = rank * sizeof(CUDA_CTYPE);
  hipMemcpy2DAsync(&(A.Xdata)[D * rank], sizeX, &(A.Ydata)[D * bdim], bdim * sizeof(CUDA_CTYPE), sizeX, M, hipMemcpyDeviceToDevice, stream);
  if (0 < rank && 0 < rdim) {
    hipMemsetAsync(A.ACdata, 0, reduc_len * M * rank * sizeof(CUDA_CTYPE), stream);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, 1, rdim, &minus_one, A.A_sr, bdim, A.Y_R_cols, bdim, &zero, A.AC_X, rank, lenA);

    while (1 < reduc_len) {
      long long len = reduc_len * rank * M;
      reduc_len = (reduc_len + 1) / 2;
      long long tail_start = reduc_len * rank * M;
      long long tail_len = len - tail_start;
      hipblasZaxpy(cublasH, tail_len, &one, &(A.ACdata)[tail_start], 1, A.ACdata, 1);
    }
    hipblasZaxpy(cublasH, M * rank, &one, A.ACdata, 1, &(A.Xdata)[D * rank], 1);
  }

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * rank;
      long long len = comm.BoxOffsets[p + 1] * rank - start;
      auto neighbor = nccl_comms.find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Xdata)[start]), &(A.Xdata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms.find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Xdata), A.Xdata, rank * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }
}

void compute_backward_substitution(deviceMatrixDesc_t A, CUDA_CTYPE* X, hipStream_t stream, hipblasHandle_t cublasH, const ColCommMPI& comm, const std::map<const MPI_Comm, ncclComm_t>& nccl_comms) {
  long long bdim = A.bdim;
  long long rank = A.rank;
  long long rdim = bdim - rank;
  long long block = bdim * bdim;

  long long D = A.diag_offset;
  long long M = comm.lenLocal();
  long long N = comm.lenNeighbors();
  long long lenA = comm.ARowOffsets[M];
  long long reduc_len = A.reducLen;

  STD_CTYPE constants[3] = { 1., 0., -1. };
  CUDA_CTYPE& one = reinterpret_cast<CUDA_CTYPE&>(constants[0]);
  CUDA_CTYPE& zero = reinterpret_cast<CUDA_CTYPE&>(constants[1]); 
  CUDA_CTYPE& minus_one = reinterpret_cast<CUDA_CTYPE&>(constants[2]); 

  if (1 < N) {
    ncclGroupStart();
    for (long long p = 0; p < (long long)comm.NeighborComm.size(); p++) {
      long long start = comm.BoxOffsets[p] * rank;
      long long len = comm.BoxOffsets[p + 1] * rank - start;
      auto neighbor = nccl_comms.find(comm.NeighborComm[p].second);
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(&(A.Xdata)[start]), &(A.Xdata)[start], len * 2, ncclDouble, comm.NeighborComm[p].first, (*neighbor).second, stream);
    }

    auto dup = nccl_comms.find(comm.DupComm);
    if (comm.DupComm != MPI_COMM_NULL)
      ncclBroadcast(const_cast<const CUDA_CTYPE*>(A.Xdata), A.Xdata, rank * N * 2, ncclDouble, 0, (*dup).second, stream);
    ncclGroupEnd();
  }

  size_t sizeX = rank * sizeof(CUDA_CTYPE);
  hipMemcpy2DAsync(&(A.Ydata)[D * bdim], bdim * sizeof(CUDA_CTYPE), &(A.Xdata)[D * rank], sizeX, sizeX, M, hipMemcpyDeviceToDevice, stream);
  if (0 < rank && 0 < rdim) {
    hipMemsetAsync(A.ACdata, 0, reduc_len * M * bdim * sizeof(CUDA_CTYPE), stream);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rdim, 1, rank, &minus_one, A.A_rs, bdim, A.X_cols, bdim, &zero, A.AC_X_R, bdim, lenA);

    while (1 < reduc_len) {
      long long len = reduc_len * bdim * M;
      reduc_len = (reduc_len + 1) / 2;
      long long tail_start = reduc_len * bdim * M;
      long long tail_len = len - tail_start;
      hipblasZaxpy(cublasH, tail_len, &one, &(A.ACdata)[tail_start], 1, A.ACdata, 1);
    }
    hipblasZaxpy(cublasH, M * bdim, &one, A.ACdata, 1, &(A.Ydata)[D * bdim], 1);
  }

  hipblasZgemmStridedBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_C, 1, bdim, bdim, &one, &(A.Ydata)[D * bdim], 1, bdim, &(A.Udata)[D * block], bdim, block, &zero, &X[A.lower_offset], 1, bdim, M);
}
