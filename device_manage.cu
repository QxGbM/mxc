
#include <factorize.cuh>

void initGpuEnvs(hipStream_t* memory_stream, hipStream_t* compute_stream, hipblasHandle_t* cublasH, hipsparseHandle_t* cusparseH, hipsolverHandle_t* cusolverH, std::map<const MPI_Comm, ncclComm_t>& nccl_comms, const std::vector<MPI_Comm>& comms, MPI_Comm world) {
  int mpi_rank, num_device;
  if (hipGetDeviceCount(&num_device) != hipSuccess)
    return;

  MPI_Comm_rank(world, &mpi_rank);
  hipSetDevice(mpi_rank % num_device);
  hipStreamCreate(memory_stream);
  hipStreamCreate(compute_stream);
  hipblasCreate(cublasH);
  hipblasSetStream(*cublasH, *compute_stream);
  hipsparseCreate(cusparseH);
  hipsparseSetStream(*cusparseH, *compute_stream);
  hipsolverDnCreate(cusolverH);
  hipsolverSetStream(*cusolverH, *compute_stream);

  long long len = comms.size();
  std::vector<ncclUniqueId> ids(len);
  std::vector<ncclComm_t> nccl_alloc(len);

  ncclGroupStart();
  for (long long i = 0; i < len; i++) {
    int rank, size;
    MPI_Comm_rank(comms[i], &rank);
    MPI_Comm_size(comms[i], &size);
    if (rank == 0)
      ncclGetUniqueId(&ids[i]);
    MPI_Bcast(reinterpret_cast<void*>(&ids[i]), sizeof(ncclUniqueId), MPI_BYTE, 0, comms[i]);
    ncclCommInitRank(&nccl_alloc[i], size, ids[i], rank);
  }
  ncclGroupEnd();

  for (long long i = 0; i < len; i++)
    nccl_comms.insert(std::make_pair(comms[i], nccl_alloc[i]));
}

void finalizeGpuEnvs(hipStream_t memory_stream, hipStream_t compute_stream, hipblasHandle_t cublasH, hipsparseHandle_t cusparseH, hipsolverHandle_t cusolverH, std::map<const MPI_Comm, ncclComm_t>& nccl_comms) {
  hipDeviceSynchronize();
  hipStreamDestroy(memory_stream);
  hipStreamDestroy(compute_stream);
  hipblasDestroy(cublasH);
  hipsparseDestroy(cusparseH);
  hipsolverDnDestroy(cusolverH);
  for (auto& c : nccl_comms)
    ncclCommDestroy(c.second);
  nccl_comms.clear();
  hipDeviceReset();
}
