#include "hip/hip_runtime.h"

#include <factorize.cuh>
#include <algorithm>
#include <numeric>
#include <tuple>

#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/transform.h>
#include <thrust/complex.h>
#include <thrust/sequence.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/async/for_each.h>

struct keysDLU {
  long long D, M, N;
  keysDLU(long long D, long long M, long long N) : D(D), M(M), N(N) {}
  __host__ __device__ long long operator()(long long y, long long x) const {
    long long diff = D + y - x;
    long long pred = (diff != 0) + (diff < 0);
    return (pred * M + y) * N + x;
  } 
};

template<class T> struct setDevicePtr {
  T* data;
  long long block;
  setDevicePtr(T* data, long long block) : data(data), block(block) {}
  __host__ __device__ T* operator()(long long i) const {
    return data + i * block;
  }
};

struct swapXY {
  long long M, B;
  swapXY(long long M, long long B) : M(M), B(B) {}
  __host__ __device__ long long operator()(long long i) const {
    long long x = i / B; long long y = i - x * B;
    long long z = y / M; long long w = y - z * M;
    return x * B + z + w * M;
  } 
};

struct conjugateDouble {
  __host__ __device__ thrust::complex<double> operator()(const thrust::complex<double>& z) const {
    return thrust::conj(z);
  }
};

template<class T> struct copyFunc {
  const T** srcs;
  T** dsts;
  long long M, B, ls, ld;
  copyFunc(long long M, long long N, const T* srcs[], long long ls, T* dsts[], long long ld) :
    srcs(srcs), dsts(dsts), M(M), B(M * N), ls(ls), ld(ld) {}
  __host__ __device__ void operator()(long long i) const {
    long long x = i / B; long long y = i - x * B;
    long long z = y / M; long long w = y - z * M;
    T e = srcs[x][z * ls + w];
    dsts[x][z * ld + w] = e;
  }
};

template<class T>
void thrust_batch_copy(hipStream_t stream, long long M, long long N, const T* srcs[], long long ls, T* dsts[], long long ld, long long batch_size) {
  auto iter = thrust::make_counting_iterator(0ll);
  auto x = thrust::async::for_each(thrust::cuda::par.on(stream), iter, iter + (M * N * batch_size), copyFunc(M, N, srcs, ls, dsts, ld));
}

void compute_factorize(hipblasHandle_t cublasH, long long bdim, long long rank, long long D, long long M, long long N, const long long ARows[], const long long ACols[], std::complex<double>* A, std::complex<double>* R, const std::complex<double>* Q) {
  long long block = bdim * bdim;
  long long lenA = ARows[M];

  hipStream_t stream;
  hipblasGetStream(cublasH, &stream);

  thrust::device_vector<long long> row_offsets(ARows, ARows + M);
  thrust::device_vector<long long> rows(lenA, 0ll);
  thrust::device_vector<long long> cols(ACols, ACols + lenA);
  thrust::device_vector<long long> keys(lenA);
  thrust::device_vector<long long> indices(lenA);
  thrust::device_vector<hipDoubleComplex*> a_ss(lenA), a_sr(lenA), a_rs(lenA), a_rr(lenA);
  thrust::device_vector<hipDoubleComplex*> u(lenA), v(lenA), v_r(lenA), b(N);

  thrust::device_vector<hipDoubleComplex> Avec(lenA * block);
  thrust::device_vector<hipDoubleComplex> Bvec(N * block);
  thrust::device_vector<hipDoubleComplex> Uvec(N * block);
  thrust::device_vector<hipDoubleComplex> Vvec(N * block);

  hipDoubleComplex* Adata = thrust::raw_pointer_cast(Avec.data());
  hipDoubleComplex* Bdata = thrust::raw_pointer_cast(Bvec.data());
  hipDoubleComplex* Udata = thrust::raw_pointer_cast(Uvec.data());
  hipDoubleComplex* Vdata = thrust::raw_pointer_cast(Vvec.data());

  hipMemcpyAsync(Udata, Q, block * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(Adata, A, block * lenA * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

  auto one_iter = thrust::make_constant_iterator(1ll);
  thrust::scatter(one_iter, one_iter + (M - 1), row_offsets.begin() + 1, rows.begin());
  thrust::inclusive_scan(rows.begin(), rows.end(), rows.begin());

  thrust::transform(rows.begin(), rows.end(), cols.begin(), keys.begin(), keysDLU(D, M, N));
  thrust::sequence(indices.begin(), indices.end(), 0);
  thrust::sort_by_key(keys.begin(), keys.end(), thrust::make_zip_iterator(rows.begin(), cols.begin(), indices.begin()));

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  auto inc_iter = thrust::make_counting_iterator(0ll);
  thrust::transform(indices.begin(), indices.end(), a_ss.begin(), setDevicePtr(Adata, block));
  thrust::transform(indices.begin(), indices.end(), a_sr.begin(), setDevicePtr(Adata + offset_SR, block));
  thrust::transform(indices.begin(), indices.end(), a_rs.begin(), setDevicePtr(Adata + offset_RS, block));
  thrust::transform(indices.begin(), indices.end(), a_rr.begin(), setDevicePtr(Adata + offset_RR, block));
  thrust::transform(cols.begin(), cols.end(), u.begin(), setDevicePtr(Udata, block));
  thrust::transform(rows.begin(), rows.end(), v.begin(), setDevicePtr(Vdata, block));
  thrust::transform(rows.begin(), rows.end(), v_r.begin(), setDevicePtr(Vdata + offset_RS, block));
  thrust::transform(inc_iter, inc_iter + N, b.begin(), setDevicePtr(Bdata, block));

  hipDoubleComplex** A_SS = thrust::raw_pointer_cast(a_ss.data());
  hipDoubleComplex** A_SR = thrust::raw_pointer_cast(a_sr.data());
  hipDoubleComplex** A_RS = thrust::raw_pointer_cast(a_rs.data());
  hipDoubleComplex** A_RR = thrust::raw_pointer_cast(a_rr.data());
  hipDoubleComplex** U = thrust::raw_pointer_cast(u.data());
  hipDoubleComplex** V = thrust::raw_pointer_cast(v.data());
  hipDoubleComplex** V_R = thrust::raw_pointer_cast(v_r.data());
  hipDoubleComplex** B = thrust::raw_pointer_cast(b.data());

  thrust::device_vector<int> Ipiv(M * bdim);
  thrust::device_vector<int> Info(M);
  int* ipiv = thrust::raw_pointer_cast(Ipiv.data());
  int* info = thrust::raw_pointer_cast(Info.data());

  long long rdim = bdim - rank;
  int info_host = 0;
  hipDoubleComplex one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);

  thrust::device_ptr<const thrust::complex<double>> u_ptr = thrust::device_ptr<const thrust::complex<double>>(reinterpret_cast<const thrust::complex<double>*>(&Udata[D * block]));
  thrust::device_ptr<thrust::complex<double>> v_ptr = thrust::device_ptr<thrust::complex<double>>(reinterpret_cast<thrust::complex<double>*>(Vdata));

  auto map = thrust::make_transform_iterator(thrust::make_counting_iterator(0ll), swapXY(bdim, block));
  thrust::gather(thrust::cuda::par.on(stream), map, map + block * M, thrust::make_transform_iterator(u_ptr, conjugateDouble()), v_ptr);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, U, bdim, A_SS, bdim, &zero, B, bdim, M);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, U, bdim, B, bdim, &zero, A_SS, bdim, M);

  hipblasZgetrfBatched(cublasH, rdim, A_RR, bdim, ipiv, info, M);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, rdim, rank, A_RR, bdim, ipiv, A_RS, bdim, &info_host, M);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, rdim, bdim, A_RR, bdim, ipiv, V_R, bdim, &info_host, M);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, rank, rdim, &minus_one, A_SR, bdim, A_RS, bdim, &one, A_SS, bdim, M);

  for (int64_t i = M; i < lenA; i += N) {
    int64_t len = std::min(lenA - i, N);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, &U[i], bdim, &A_SS[i], bdim, &zero, B, bdim, len);
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, bdim, bdim, bdim, &one, &V[i], bdim, B, bdim, &zero, &A_SS[i], bdim, len);
  }
  hipStreamSynchronize(stream);

  hipMemcpy(A, Adata, block * lenA * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&R[block * D], Vdata, block * M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}
