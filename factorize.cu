#include "hip/hip_runtime.h"

#include <factorize.cuh>
#include <comm-mpi.hpp>
#include <algorithm>
#include <numeric>
#include <tuple>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/transform.h>
#include <thrust/complex.h>
#include <thrust/sequence.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/scan.h>

struct keysDLU {
  long long D, M, N;
  keysDLU(long long D, long long M, long long N) : D(D), M(M), N(N) {}
  __host__ __device__ long long operator()(long long y, long long x) const {
    long long diff = D + y - x;
    long long pred = (diff != 0) + (diff < 0);
    return (pred * M + y) * N + x;
  }
};

template<class T> struct setDevicePtr {
  T* data;
  long long block, brows;
  setDevicePtr(T* data, long long block, long long brows = 0) : data(data), block(block), brows(brows) {}
  __host__ __device__ T* operator()(long long i) const {
    return data + i * block;
  }
  __host__ __device__ T* operator()(long long y, long long x) const {
    return data + (y + x * brows) * block;
  }
};

struct swapXY {
  long long M, B;
  swapXY(long long M, long long B) : M(M), B(B) {}
  __host__ __device__ long long operator()(long long i) const {
    long long x = i / B; long long y = i - x * B;
    long long z = y / M; long long w = y - z * M;
    return x * B + z + w * M;
  } 
};

struct conjugateDouble {
  __host__ __device__ thrust::complex<double> operator()(const thrust::complex<double>& z) const {
    return thrust::conj(z);
  }
};

template<class T> struct copyFunc {
  const T** srcs;
  T** dsts;
  long long M, B, ls, ld;
  copyFunc(long long M, long long N, const T* srcs[], long long ls, T* dsts[], long long ld) :
    srcs(srcs), dsts(dsts), M(M), B(M * N), ls(ls), ld(ld) {}
  __host__ __device__ void operator()(long long i) const {
    long long x = i / B; long long y = i - x * B;
    long long z = y / M; long long w = y - z * M;
    T e = srcs[x][z * ls + w];
    dsts[x][z * ld + w] = e;
  }
};

void compute_factorize(hipblasHandle_t cublasH, long long bdim, long long rank, long long D, long long M, long long N, const long long ARows[], const long long ACols[], std::complex<double>* A, std::complex<double>* R, const std::complex<double>* Q, const ColCommMPI& comm) {
  long long block = bdim * bdim;
  long long lenA = ARows[M];
  
  hipStream_t stream;
  hipblasGetStream(cublasH, &stream);

  thrust::device_vector<long long> row_offsets(ARows, ARows + M);
  thrust::device_vector<long long> rows(lenA, 0ll);
  thrust::device_vector<long long> cols(ACols, ACols + lenA);
  thrust::device_vector<long long> dist_cols(lenA);
  thrust::device_vector<long long> keys(lenA);
  thrust::device_vector<long long> indices(lenA);
  thrust::device_vector<hipDoubleComplex*> a_ss(lenA), a_sr(lenA), a_rs(lenA), a_rr(lenA);
  thrust::device_vector<hipDoubleComplex*> u(lenA), v(lenA), u_r(M), v_r(M), b(N), b_cols(lenA), b_i_cols(lenA);
  thrust::device_vector<hipDoubleComplex*> a_sr_rows(lenA), acc(lenA), acc_final(M);

  thrust::device_vector<hipDoubleComplex> Avec(lenA * block);
  thrust::device_vector<hipDoubleComplex> Bvec(N * block);
  thrust::device_vector<hipDoubleComplex> Uvec(N * block);
  thrust::device_vector<hipDoubleComplex> Vvec(M * block);
  
  thrust::device_vector<int> Ipiv(M * bdim);
  thrust::device_vector<int> Info(M);
  std::vector<long long> Bsizes(N, block);
  comm.dataSizesToNeighborOffsets(Bsizes.data());

  auto inc_iter = thrust::make_counting_iterator(0ll);
  auto one_iter = thrust::make_constant_iterator(1ll);
  auto rwise_diag_iter = thrust::make_permutation_iterator(indices.begin(), rows.begin());

  hipDoubleComplex* Adata = thrust::raw_pointer_cast(Avec.data());
  hipDoubleComplex* Udata = thrust::raw_pointer_cast(Uvec.data());
  hipDoubleComplex* Vdata = thrust::raw_pointer_cast(Vvec.data());
  hipDoubleComplex* Bdata = thrust::raw_pointer_cast(Bvec.data());

  hipMemcpyAsync(Udata, Q, block * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(Adata, A, block * lenA * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

  thrust::scatter(one_iter, one_iter + (M - 1), row_offsets.begin() + 1, rows.begin()); 
  thrust::inclusive_scan(rows.begin(), rows.end(), rows.begin());
  thrust::exclusive_scan_by_key(rows.begin(), rows.end(), one_iter, dist_cols.begin(), 0ll);

  thrust::transform(rows.begin(), rows.end(), cols.begin(), keys.begin(), keysDLU(D, M, N));
  thrust::sequence(indices.begin(), indices.end(), 0);
  thrust::sort_by_key(keys.begin(), keys.end(), thrust::make_zip_iterator(rows.begin(), cols.begin(), dist_cols.begin(), indices.begin()));

  long long reduc_len = 1ll + thrust::reduce(dist_cols.begin(), dist_cols.end(), 0ll, thrust::maximum<long long>());
  thrust::device_vector<hipDoubleComplex> ACvec(reduc_len * M * rank * rank, make_hipDoubleComplex(0., 0.));
  hipDoubleComplex* ACdata = thrust::raw_pointer_cast(ACvec.data());

  long long offset_SR = rank * bdim, offset_RS = rank, offset_RR = rank * (bdim + 1);
  thrust::transform(indices.begin(), indices.end(), a_ss.begin(), setDevicePtr(Adata, block));
  thrust::transform(indices.begin(), indices.end(), a_sr.begin(), setDevicePtr(Adata + offset_SR, block));
  thrust::transform(indices.begin(), indices.end(), a_rs.begin(), setDevicePtr(Adata + offset_RS, block));
  thrust::transform(indices.begin(), indices.end(), a_rr.begin(), setDevicePtr(Adata + offset_RR, block));
  thrust::transform(cols.begin(), cols.end(), u.begin(), setDevicePtr(Udata, block));
  thrust::transform(cols.begin(), cols.begin() + M, u_r.begin(), setDevicePtr(Udata + offset_SR, block));
  thrust::transform(rows.begin(), rows.end(), v.begin(), setDevicePtr(Vdata, block));
  thrust::transform(inc_iter, inc_iter + M, v_r.begin(), setDevicePtr(Vdata + offset_RS, block));

  thrust::transform(inc_iter, inc_iter + N, b.begin(), setDevicePtr(Bdata, block));
  thrust::transform(cols.begin(), cols.end(), b_cols.begin(), setDevicePtr(Bdata, block));
  thrust::transform(cols.begin(), cols.end(), b_i_cols.begin(), setDevicePtr(Bdata + offset_SR, block));
  thrust::transform(rwise_diag_iter, rwise_diag_iter + lenA, a_sr_rows.begin(), setDevicePtr(Adata + offset_SR, block));
  thrust::transform(rows.begin(), rows.end(), dist_cols.begin(), acc.begin(), setDevicePtr(ACdata, rank * rank, M));
  thrust::transform(inc_iter, inc_iter + M, acc_final.begin(), setDevicePtr(ACdata, rank * rank));

  hipDoubleComplex** A_SS = thrust::raw_pointer_cast(a_ss.data());
  hipDoubleComplex** A_SR = thrust::raw_pointer_cast(a_sr.data());
  hipDoubleComplex** A_RS = thrust::raw_pointer_cast(a_rs.data());
  hipDoubleComplex** A_RR = thrust::raw_pointer_cast(a_rr.data());
  hipDoubleComplex** U = thrust::raw_pointer_cast(u.data());
  hipDoubleComplex** U_R = thrust::raw_pointer_cast(u_r.data());
  hipDoubleComplex** V = thrust::raw_pointer_cast(v.data());
  hipDoubleComplex** V_R = thrust::raw_pointer_cast(v_r.data());
  hipDoubleComplex** B = thrust::raw_pointer_cast(b.data());
  hipDoubleComplex** B_Cols = thrust::raw_pointer_cast(b_cols.data());
  hipDoubleComplex** B_I_Cols = thrust::raw_pointer_cast(b_i_cols.data());
  hipDoubleComplex** A_SR_Rows = thrust::raw_pointer_cast(a_sr_rows.data());
  hipDoubleComplex** ACC = thrust::raw_pointer_cast(acc.data());
  hipDoubleComplex** ACC_Final = thrust::raw_pointer_cast(acc_final.data());

  int* ipiv = thrust::raw_pointer_cast(Ipiv.data());
  int* info = thrust::raw_pointer_cast(Info.data());

  long long rdim = bdim - rank;
  int info_host = 0;
  hipDoubleComplex one = make_hipDoubleComplex(1., 0.), zero = make_hipDoubleComplex(0., 0.), minus_one = make_hipDoubleComplex(-1., 0.);

  thrust::device_ptr<const thrust::complex<double>> u_ptr = thrust::device_ptr<const thrust::complex<double>>(reinterpret_cast<const thrust::complex<double>*>(&Udata[D * block]));
  thrust::device_ptr<thrust::complex<double>> v_ptr = thrust::device_ptr<thrust::complex<double>>(reinterpret_cast<thrust::complex<double>*>(Vdata));

  auto map = thrust::make_transform_iterator(thrust::make_counting_iterator(0ll), swapXY(bdim, block));
  thrust::gather(thrust::cuda::par.on(stream), map, map + block * M, thrust::make_transform_iterator(u_ptr, conjugateDouble()), v_ptr);

  hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, U, bdim, A_SS, bdim, &zero, B, bdim, M);
  hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, U, bdim, B, bdim, &zero, A_SS, bdim, M);

  hipblasZgetrfBatched(cublasH, rdim, A_RR, bdim, ipiv, info, M);
  hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, rdim, bdim, A_RR, bdim, ipiv, V_R, bdim, &info_host, M);

  if (0 < rank) {
    hipblasZgetrsBatched(cublasH, HIPBLAS_OP_N, rdim, rank, A_RR, bdim, ipiv, A_RS, bdim, &info_host, M);

    for (long long i = M; i < lenA; i += N) {
      long long len = std::min(lenA - i, N);
      hipblasZgemmBatched(cublasH, HIPBLAS_OP_C, HIPBLAS_OP_T, bdim, bdim, bdim, &one, &U[i], bdim, &A_SS[i], bdim, &zero, B, bdim, len);
      hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, bdim, bdim, bdim, &one, &V[i], bdim, B, bdim, &zero, &A_SS[i], bdim, len);
    }
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, rank, rdim, &minus_one, A_SR_Rows, bdim, A_RS, bdim, &one, A_SS, bdim, lenA);

    thrust::for_each(thrust::cuda::par.on(stream), inc_iter, inc_iter + (rdim * rank * M), copyFunc(rdim, rank, const_cast<const hipDoubleComplex**>(A_RS), bdim, &B[D], bdim));
    hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rdim, rdim, bdim, &one, V_R, bdim, U_R, bdim, &zero, B_I_Cols, bdim, M);
    hipMemcpyAsync(&R[D * block], &Bdata[D * block], M * block * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost, stream);
    thrust::for_each(thrust::cuda::par.on(stream), inc_iter, inc_iter + (rank * rank * M), copyFunc(rank, rank, const_cast<const hipDoubleComplex**>(A_SS), bdim, ACC, rank));

    comm.neighbor_bcast(R, Bsizes.data());
    hipMemcpyAsync(Bdata, R, N * block * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);

    if (M < lenA)
      hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, rank, rdim, &minus_one, &A_SR[M], bdim, &B_Cols[M], bdim, &one, &A_SS[M], bdim, lenA - M);

    for (long long i = M; i < lenA; i += N) {
      long long len = std::min(lenA - i, N);
      hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, rdim, rank, rdim, &one, &B_I_Cols[i], bdim, &A_SR[i], bdim, &zero, B, bdim, len);
      hipblasZgemmBatched(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, rank, rank, rdim, &minus_one, &A_SR[i], bdim, B, bdim, &zero, &ACC[i], rank, len);
    }

    while (1 < reduc_len) {
      long long len = reduc_len * rank * rank * M;
      reduc_len = (reduc_len + 1) / 2;
      long long tail_start = reduc_len * rank * rank * M;
      long long tail_len = len - tail_start;
      hipblasZaxpy(cublasH, tail_len, &one, &ACdata[tail_start], 1, ACdata, 1);
    }
    thrust::for_each(thrust::cuda::par.on(stream), inc_iter, inc_iter + (rank * rank * M), copyFunc(rank, rank, const_cast<const hipDoubleComplex**>(ACC_Final), rank, A_SS, bdim));
  }
  hipStreamSynchronize(stream);

  hipMemcpy(A, Adata, block * lenA * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  hipMemcpy(&R[block * D], Vdata, block * M * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}
